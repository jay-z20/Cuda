#include<hip/hip_runtime.h>


#include<stdio.h>
#include<stdlib.h>


#define BDIMY 32
#define BDIMX 32

void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)( rand() & 0xFF ) / 10.0f; //100.0f;
    }

    return;
}


void checkResult(float *hostRef, float *gpuRef, const int size, int showme)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < size; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }

        if (showme && i > size / 2 && i < size / 2 + 5)
        {
            // printf("%dth element: host %f gpu %f\n",i,hostRef[i],gpuRef[i]);
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}


void transposeHost(float *out, float *in, const int nx, const int ny)
{
    for( int iy = 0; iy < ny; ++iy)
    {
        for( int ix = 0; ix < nx; ++ix)
        {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}



// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
// �߳̿� x,y = 32,8
__global__ void transposeCoalesced(float *out, const float *in,const int nx,const int ny)
{
  __shared__ float tile[BDIMY][BDIMX+1]; // 32x32
    
  int x = blockIdx.x * BDIMX + threadIdx.x;
  int y = blockIdx.y * BDIMY + threadIdx.y;
  int width =nx;
    // ÿ���̼߳��� 4��
  for (int j = 0; j < BDIMX; j += 8)
     tile[threadIdx.y+j][threadIdx.x] = in[(y+j)*width + x]; // д�����ڴ�û�д洢���ͻֻʹ���� 1������

  __syncthreads();

  x = blockIdx.y * BDIMX + threadIdx.x;  // transpose block offset
  y = blockIdx.x * BDIMX + threadIdx.y;
  width = ny;

  for (int j = 0; j < BDIMX; j += 8)
     out[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j]; // һ���߳���
}
   

// <<CUDA C���Ȩ��ָ��>> ���� nx!=ny �������ȷ
// out[ix][iy] = in[iy][ix]
__global__ void transposeSmem(float *out,float *in,const int nx,const int ny){
    
    __shared__ float tile[BDIMY][BDIMX];
    
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix;
    

    // �߳̿��ڵ����
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y; // ���� ת�ú������
    unsigned int icol = bidx % blockDim.y;

    ix = blockIdx.y * blockDim.y + icol;
    iy = blockIdx.x * blockDim.x + irow;
    
    unsigned int to = iy * ny + ix;

    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[ti];
        __syncthreads();
        out[to] = tile[icol][irow];
    }
    
}

bool check(float *out,float *res,const int nx,const int ny){
    for(int i=0;i<nx;i++){
        for (int j = 0; j < ny; j++){
            if (out[i*ny+j]!=res[i*ny+j]){
                return false;
            }
        }
    }
    return true;
}


int main(){
    const int nx = 1024;
    const int ny = 2048;

    const size_t N = nx*ny;
    const size_t nBytes = N*sizeof(float);

    float *a = (float*) malloc(nBytes);
    float *out = (float*) malloc(nBytes);
    float *res = (float*)malloc(nBytes);

    float *d_a, *d_out;
    hipMalloc((float**)&d_a,nBytes);
    hipMalloc((float**)&d_out,nBytes);
    
    // ��ʼ������
    initialData(a,N);

    transposeHost(res,a,nx,ny);
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_a,a,nBytes,hipMemcpyHostToDevice);
    dim3 Block(32,8);
    dim3 Grid((nx+32-1)/32,(ny+32-1)/32);
    hipEventRecord(start);
    /// transposeUnroll4Row  transposeUnroll4Col
    //transposeSmem<<<Grid,Block>>>(d_out,d_a,nx,ny);
    transposeCoalesced<<<Grid,Block>>>(d_out,d_a,nx,ny);
    hipEventRecord(stop);
    // �ȴ� stop event ���
    hipEventSynchronize(stop);

    hipMemcpy(out,d_out,nBytes,hipMemcpyDeviceToHost);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Time: %f ms\n",milliseconds);
    printf("Bandwidth (GB/s): %f\n",(N*4 + N*4)/milliseconds/1e6);
    checkResult(res,out,N,1);
    if(check(out,res,nx,ny))
        printf("the ans is right\n");
    else
        printf("the ans is wrong\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
    free(res);
    return 0;
}
