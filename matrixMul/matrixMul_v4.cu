
#include<stdio.h>
#include<stdlib.h>

#include<hip/hip_runtime.h>
#include<hipblas.h>



#define BLOCK_SIZE 16

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template<
    const int BLOCK_SIZE_M,  // width of block of C that each thread block calculate 128
    const int BLOCK_SIZE_K,  // height of block of A that each thread block load into shared memory 8
    const int BLOCK_SIZE_N,  // height of block of C that each thread block calculate 128
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate 8
    const int THREAD_SIZE_X  // width of block of C that each thread calculate 8
>
__global__ void matrixMul(float* A,float* B,float* C,const int M,
    const int K,const int N){
    
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int bszx = BLOCK_SIZE_N / THREAD_SIZE_X; // 128/8=16
    const int bszy = BLOCK_SIZE_M / THREAD_SIZE_Y; // 128/8=16
    const int THREAD_NUM_PER_BLOCK = bszx * bszy; // 256

    // thread id
    const int tid = ty * bszx + tx;

    __shared__ float As[2][BLOCK_SIZE_M][BLOCK_SIZE_K+1];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];

    // register for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // register for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    
    // 一个 tile 需要的线程数,float4 所以需要 除以 4
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4; // 8/4=2 tile A 每行需要 2个线程
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4; // 128/4=32 tile B 每行需要 32 个线程

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;  // 线程 在 tile A 加载数据的起始行
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW; // 线程 在 tile B 加载数据的起始行

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; // 线程在 tile A 中加载数据的列
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4; // 线程在 tile B 中加载数据的列

     // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW; // A tile 的行跨度 256/2=128
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW; // B tile 的行跨度
    const int idx = A_TILE_ROW_START * BLOCK_SIZE_K + A_TILE_COL; // A tile 行号x宽 + 列
    const int idx_ = ty * THREAD_SIZE_Y * BLOCK_SIZE_K; // 一个线程计算结果 8x8，

    // Prefetching
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int r = (idx + i * BLOCK_SIZE_K) / (BLOCK_SIZE_K + 1);
        int c = (idx + i * BLOCK_SIZE_K) % (BLOCK_SIZE_K + 1);
        FETCH_FLOAT4(As[0][r][c]) = FETCH_FLOAT4(A[OFFSET(
                BLOCK_SIZE_M * by + A_TILE_ROW_START + i, // row
                A_TILE_COL , // col
                K )]);
    }

    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL + BLOCK_SIZE_N * bx, // col
                N )]);
    }

    __syncthreads();

    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        int r = (idx_ + thread_y * BLOCK_SIZE_K ) / (BLOCK_SIZE_K + 1);
        int c = (idx_ + thread_y * BLOCK_SIZE_K ) % (BLOCK_SIZE_K + 1);
        frag_a[0][thread_y] = As[0][r][c];
    }

    // load B from shared memory to register
    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    int write_stage_idx = 1;
    int load_stage_idx = 0;
    for (int tile_idx = BLOCK_SIZE_K ; tile_idx <= K ; tile_idx += BLOCK_SIZE_K){ // tile_idx < K 修改为 tile_idx <= K，为了能够计算足够次数
        // load A from global memory to shared memory
        // 一个线程取 float4 4个数据，一个线程块 16x16=256 个线程，取 256x4=128x8 的数据块
        // Prefetching 加载下一次需要计算的数据
        if (tile_idx < K)
        {
             #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int r = (idx + i * BLOCK_SIZE_K) / (BLOCK_SIZE_K + 1);
                int c = (idx + i * BLOCK_SIZE_K) % (BLOCK_SIZE_K + 1);
                FETCH_FLOAT4(As[write_stage_idx][r][c]) = FETCH_FLOAT4(A[OFFSET(
                        BLOCK_SIZE_M * by + A_TILE_ROW_START + i, // row
                        A_TILE_COL + tile_idx, // col
                        K )]);
            }

            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                        tile_idx + B_TILE_ROW_START + i, // row
                        B_TILE_COL + BLOCK_SIZE_N * bx, // col
                        N )]);
            }
        
            __syncthreads();
        }
        
         // compute c
        #pragma unroll
        for (int k = 1; k <= BLOCK_SIZE_K; ++ k) { //  k <= BLOCK_SIZE_K 修改为  k <= BLOCK_SIZE_K
            // load A from shared memory to register
            if ( k < BLOCK_SIZE_K)
            {
                #pragma unroll
                for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                    int r = (idx_ + thread_y * BLOCK_SIZE_K + k) / (BLOCK_SIZE_K + 1);
                    int c = (idx_ + thread_y * BLOCK_SIZE_K + k) % (BLOCK_SIZE_K + 1);
                    frag_a[k%2][thread_y] = As[load_stage_idx][r][c];
                }

                // load B from shared memory to register
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                    FETCH_FLOAT4(frag_b[k%2][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx][k][THREAD_SIZE_X * tx + thread_x]);
                }
            }
            
            
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[(k-1)%2][thread_y] * frag_b[(k-1)%2][thread_x];
                }
            }
            
        }
        __syncthreads();
        write_stage_idx = write_stage_idx ^ 1;
        load_stage_idx = load_stage_idx ^ 1;
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
            C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)] = accum[thread_y][thread_x];
        }
    }
}

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}



bool check(float *out,float *res,const int nx,const int ny){
    for(int i=0;i<nx;i++){
        for (int j = 0; j < ny; j++){
            if (out[i*ny+j]!=res[i*ny+j]){
                printf("i: %d j:%d\n",i,j);
                printf("out: %.2f res:%.2f\n",out[i*ny+j],res[i*ny+j]);
                return false;
            }
        }
    }
    return true;
}


int main(int argc,char **argv){
    if (argc != 4)
    {
        printf("usage: main.exe [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));

    double flopsPerMatrixMul = 2.0 * M * N * K;

    // initialize host memory
    srand(2022);
    randomInit(h_A, M * K);
    randomInit(h_B, K * N);

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));

    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;

    float milliseconds = 0;
    dim3 Block(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 Grid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    //
    checkCudaErrors(hipEventRecord(start));
    matrixMul<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X>
    <<<Grid,Block>>>(d_A,d_B,d_C,M,K,N);

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));
    printf("------ V4 ------\n");
    printf("Time: %f ms\n",milliseconds);
    printf("Performance (GFlop/s): %.2f\n",(flopsPerMatrixMul)/milliseconds/1e6);
    //cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipEventRecord(start));


// cublasStatus_t cublasSgemm(cublasHandle_t handle,
//                            cublasOperation_t transa, cublasOperation_t transb,
//                            int m, int n, int k,
//                            const float           *alpha,
//                            const float           *A, int lda,
//                            const float           *B, int ldb,
//                            const float           *beta,
//                            float           *C, int ldc)
            // lda
            // op = CUBLAS_OP_N 时：m
            // op = CUBLAS_OP_T 时：k
            // ldb
            // op = CUBLAS_OP_N 时：k
            // op = CUBLAS_OP_T 时：n
            // ldc: m
            
    hipblasSgemm (blas_handle,
            HIPBLAS_OP_N,  
            HIPBLAS_OP_N,  
            M,  
            N,  
            K,  
            &alpha,  
            d_B,  
            N,     
            d_A, 
            K,  
            &beta, 
            d_C,  
            N   
        );

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    checkCudaErrors(hipMemcpy(h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    hipblasDestroy(blas_handle); 

    printf("------ cublas ------\n");
    printf("Time: %f ms\n",milliseconds);
    printf("Performance (GFlop/s): %.2f\n",(flopsPerMatrixMul)/milliseconds/1e6);

    if(check(h_C,h_C1,N,M))
        printf("the ans is right\n");
    else
        printf("the ans is wrong\n");

    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
    
    return 0;
}


