#include<hip/hip_runtime.h>


#include<stdio.h>


#define THREAD_PER_BLOCK 256

__global__ void reduce4(float* d_in,float* d_out,unsigned int n){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x*8 + threadIdx.x;
    if (idx >=n) return;
    
    float *idata = d_in + blockIdx.x * blockDim.x*8;
    if (idx + 7*blockDim.x < n)
    {
        int a1 = d_in[idx];
        int a2 = d_in[idx+blockDim.x];
        int a3 = d_in[idx+2*blockDim.x];
        int a4 = d_in[idx+3*blockDim.x];
        int b1 = d_in[idx+4*blockDim.x];
        int b2 = d_in[idx+5*blockDim.x];
        int b3 = d_in[idx+6*blockDim.x];
        int b4 = d_in[idx+7*blockDim.x];
        d_in[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
     __syncthreads();

    for (int stride = blockDim.x/2; stride >32; stride>>=1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }
    if (tid < 32)
    {
       volatile float *vmem = idata;
       vmem[tid] += vmem[tid + 32];
       vmem[tid] += vmem[tid + 16];
       vmem[tid] += vmem[tid + 8];

       vmem[tid] += vmem[tid + 4];
       vmem[tid] += vmem[tid + 2];
       vmem[tid] += vmem[tid + 1];
    }
    
    if (tid==0) d_out[blockIdx.x] = idata[0];
}


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}


int main(){
    const int N = 32*1024*1024;
    int block_num = N / THREAD_PER_BLOCK/4;


    float *a = (float*) malloc(N*sizeof(float));
    float *out = (float*) malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    float *res = (float*)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    float *d_a, *d_out;
    hipMalloc((void**)&d_a,N*sizeof(float));
    hipMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));

    for (size_t i = 0; i < N; i++)
    {
        a[i] = 1;
    }
    
    for (size_t i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (size_t j = 0; j < THREAD_PER_BLOCK*8; j++)
        {
            cur += a[i*THREAD_PER_BLOCK +j];
        }
        res[i] = cur;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);
    dim3 Grid(N/THREAD_PER_BLOCK/8,1);
    dim3 Block(THREAD_PER_BLOCK,1);
    hipEventRecord(start);
    ///
    reduce4<<<Grid,Block>>>(d_a,d_out,N);
    hipEventRecord(stop);
    // 等待 stop event 完成
    hipEventSynchronize(stop);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Time: %f ms\n",milliseconds);
    printf("Bandwidth (GB/s): %f\n",(N*4 + block_num*4)/milliseconds/1e6);

    if(check(out,res,block_num))printf("the ans is right\n");
        else{
            printf("the ans is wrong\n");
            for(int i=0;i<block_num;i++){
                printf("%lf ",out[i]);
            }
            printf("\n");
        }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
    free(res);
    return 0;
}
