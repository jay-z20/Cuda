#include<hip/hip_runtime.h>


#include<stdio.h>


#define THREAD_PER_BLOCK 256

__global__ void reduce2(float* d_in,float* d_out,unsigned int n){
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >=n) return;
    
    float *idata = d_in + blockIdx.x * blockDim.x;
    for (int stride = blockDim.x/2; stride >0; stride>>=1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid+stride];
        }
        __syncthreads();
    }
    if (tid==0) d_out[blockIdx.x] = idata[0];
}


bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}


int main(){
    const int N = 32*1024*1024;
    int block_num = N / THREAD_PER_BLOCK/4;


    float *a = (float*) malloc(N*sizeof(float));
    float *out = (float*) malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    float *res = (float*)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    float *d_a, *d_out;
    hipMalloc((void**)&d_a,N*sizeof(float));
    hipMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));

    for (size_t i = 0; i < N; i++)
    {
        a[i] = 1;
    }
    
    for (size_t i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (size_t j = 0; j < THREAD_PER_BLOCK; j++)
        {
            cur += a[i*THREAD_PER_BLOCK +j];
        }
        res[i] = cur;
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);
    dim3 Grid(N/THREAD_PER_BLOCK,1);
    dim3 Block(THREAD_PER_BLOCK,1);
    hipEventRecord(start);
    ///
    reduce2<<<Grid,Block>>>(d_a,d_out,N);
    hipEventRecord(stop);
    // 等待 stop event 完成
    hipEventSynchronize(stop);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Time: %f ms\n",milliseconds);
    printf("Bandwidth (GB/s): %f\n",(N*4 + block_num*4)/milliseconds/1e6);

    if(check(out,res,block_num))printf("the ans is right\n");
        else{
            printf("the ans is wrong\n");
            for(int i=0;i<block_num;i++){
                printf("%lf ",out[i]);
            }
            printf("\n");
        }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
    free(res);
    return 0;
}
